#include "hip/hip_runtime.h"
#define num_blocks 16
#define num_threads 256
#define shared_memory_usage 0

#define MaxPixel 10
#define StreamNum 4
#define BufferNum 4
#define CONSTANT_SIZE 1000 
#define NIFO 4
#define nIFO 3
#define XIFO 4
#define CUDA_CHECK(value) {                                             \
    hipError_t _m_cudaStat = value;                                    \
    if (_m_cudaStat != hipSuccess) {                                   \
        fprintf(stderr, "Error %s at line %d in file %s\n",             \
                hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);   \
        exit(1); }}



#include "/home/hpc/cWB/TEST/S6A_BKG_LF_L1H1V1_2G_SUPERCLUSTER_run1a_bench2/macro/test/test_struct.h"
#include "/home/hpc/cWB/TEST/S6A_BKG_LF_L1H1V1_2G_SUPERCLUSTER_run1a_bench2/macro/test/main.cuh"
#include<fstream>
#include<iostream>
using namespace std;

__constant__ float constEn, constEs;				// GPU constant memory
__constant__ size_t constV[CONSTANT_SIZE], constV4[CONSTANT_SIZE], consttsize[CONSTANT_SIZE];
int main(void)
{
	struct pre_data pre_gpu_data[BufferNum];	// store the data before gpu calculation
	struct post_data post_gpu_data[StreamNum];	// store the data transfer from gpu
	struct skyloop_output skyloop_output[StreamNum];	// store the skyloop_output data
	struct other skyloop_other[StreamNum];		// store the data which is not output
	
	ifstream file1("../../skyloop_mm");
	ifstream file2("../../skyloop_ml");
	ifstream file3("../../skyloop_eTD");
	ifstream file4("../../skyloop_inputV");
	
	short temp1;
	short temp2;
	
	int Tmax, V4max, eTDDim, Lsky;
	int FileDim;
	int alloced_gpu = 0;
	int K, k, i;
	size_t V, V4, tsize;
	size_t *V_array, *V4_array, *tsize_array;
	int *k_sortArray;
	size_t k_array[StreamNum][MaxPixel];
	float En, Es;
	int etddim;
	size_t ptr;					// indicate the pointer
	int pixelCount;					// indicate the number of pixels that are send to the stream
	size_t etddim_array[StreamNum];
	size_t alloced_V4_array[StreamNum];
	int pixel_array[StreamNum];
	size_t alloced_V4;
	int CombineSize;
	Tmax = 219;
	V4max = 100;
	Lsky = 3072;
	eTDDim = Tmax * V4max;
	FileDim = 219*300;
	CombineSize = V4max / 2;
	allocate_cpu_mem(pre_gpu_data, post_gpu_data, eTDDim, V4max, Lsky);
	allocate_gpu_mem(skyloop_output, skyloop_other, eTDDim, V4max, Lsky);

	hipStream_t stream[StreamNum];			// define the stream
	for(int i=0; i<StreamNum; i++)			
		CUDA_CHECK(hipStreamCreate(&stream[i]));	// create the new stream

	for(int l=0; l<Lsky; l++)
	{
		for(int j=0; j<NIFO; j++)
		{
			file2>>temp2;
			int mlptr;
			mlptr = j * Lsky;
			pre_gpu_data[0].other_data.ml_mm[mlptr + l] = temp2;
		}
		file1>>temp1;
		pre_gpu_data[0].other_data.ml_mm[NIFO * Lsky + l] = temp1;
	}

	hipMemcpyAsync(skyloop_other[0].ml_mm, pre_gpu_data[0].other_data.ml_mm, (1+NIFO) * Lsky * sizeof(short), hipMemcpyHostToDevice, stream[0] );
	
	K = 115;
	V_array = (size_t*)malloc(sizeof(size_t) * K);
	V4_array = (size_t*)malloc(sizeof(size_t) * K);
	tsize_array = (size_t*)malloc(sizeof(size_t) * K);
	k_sortArray = (int*)malloc(sizeof(int) * K);
	 
	clock_t start[10], finish[10];
	double d[10];
	for(int z=0; z<10; z++)
		d[z] = 0;
	for(int k=0; k<K; k++)
	{
		file4>>V;
		file4>>V4;
		file4>>tsize;	
		file4>>En;
		file4>>Es;
		if(k==0)
		{
			hipMemcpyToSymbol(HIP_SYMBOL(constEn), &En, sizeof(float));
			hipMemcpyToSymbol(HIP_SYMBOL(constEs), &Es, sizeof(float));
		}
		V_array[k] = V;
		V4_array[k] = V4;
		tsize_array[k] = tsize;
		k_sortArray[k] = k;
				
	}
	
	hipMemcpyToSymbol(HIP_SYMBOL(constV), V_array, sizeof(size_t) * K);
//	hipMemcpyToSymbol(HIP_SYMBOL(constV4), V4_array, sizeof(size_t) * K);
	hipMemcpyToSymbol(HIP_SYMBOL(consttsize), tsize_array, sizeof(size_t) * K);
	
	QuickSort(V_array, k_sortArray, 0, K-1);
	for(int i=0; i<K; i++)
		cout<<"K = "<<k_sortArray[i]<<" V = "<<V_array[k_sortArray[i]]<<endl;

	start[0] = clock();
//	FILE *fpt = fopen("skyloop_myeTD", "a");
	alloced_V4 = 0;
	ptr = MaxPixel;
	pixelCount = 0;
	for(int count=0; count<K; count++)
	{
		start[1] = clock();
		k = k_sortArray[count];
		i = alloced_gpu;
		etddim = V4_array[k] * tsize_array[k];
		alloced_V4 += V4_array[k];
		if(alloced_V4<=V4max && (alloced_V4 + pixelCount + 1)<=(V4max+1))
		{
			string t;
			file3.seekg(0, ios_base::beg);
			for(int j=0; j<k*FileDim; j++)
				getline(file3, t);
			for(int l=0; l<etddim; l++)
			{
				file3>>pre_gpu_data[alloced_gpu].other_data.eTD[ptr + l];
				file3>>pre_gpu_data[alloced_gpu].other_data.eTD[ptr + l + etddim];
				file3>>pre_gpu_data[alloced_gpu].other_data.eTD[ptr + l + 2 * etddim];
				file3>>pre_gpu_data[alloced_gpu].other_data.eTD[ptr + l + 3 * etddim];
			}
			ptr += NIFO*etddim;
			k_array[i][pixelCount] = k+1;
			pre_gpu_data[i].other_data.eTD[pixelCount] = k+1;
			post_gpu_data[i].other_data.k[pixelCount] = k+1;
			post_gpu_data[i].other_data.V4[pixelCount] = V4_array[k];
			post_gpu_data[i].other_data.tsize[pixelCount] = tsize_array[k];
			pixelCount++;
		}
		finish[1] = clock();
		d[1] += (double)(finish[1] - start[1]);
		if( pixelCount<MaxPixel && alloced_V4<CombineSize )	continue;
		else if(alloced_V4>V4max || (alloced_V4 + pixelCount)>(V4max+1))
		{
			alloced_V4 -= V4_array[k];
			count--;
		}
		cout<<"list"<<endl;
		for(int z=0; z<pixelCount; z++)
			cout<<"k = "<<pre_gpu_data[i].other_data.eTD[z]-1<<endl;
		etddim_array[i] = ptr;
		alloced_V4_array[i] = alloced_V4;
		pixel_array[i] = pixelCount;
		alloced_gpu++;
		if(alloced_gpu == StreamNum)		// if all streams' data have been assigned
		{
			start[2] = clock();
			push_work_into_gpu(pre_gpu_data, post_gpu_data, skyloop_output, skyloop_other, alloced_V4_array, etddim_array, k_array, Lsky, pixel_array, StreamNum, stream);
			for(int i=0; i<StreamNum; i++)				// wait for all commands in the stream to complete
				CUDA_CHECK(hipStreamSynchronize(stream[i]));
			alloced_gpu = 0;
			finish[2] = clock();
			d[2] += (double)(finish[2] - start[2])/CLOCKS_PER_SEC;
			for(int j=0; j<StreamNum; j++)
				for(int i=0; i<pixel_array[j]; i++)
				{
					k_array[j][i] = 0;
					post_gpu_data[j].other_data.k[i] = 0;	
					post_gpu_data[j].other_data.V4[i] = 0;
					post_gpu_data[j].other_data.tsize[i] = 0;
				}
		}
		//clear
		ptr = MaxPixel;
		pixelCount = 0;
		alloced_V4 = 0;
	}
	//fclose(fpt);
	if(alloced_gpu != 0)
	{
		start[2] = clock();
		push_work_into_gpu(pre_gpu_data, post_gpu_data, skyloop_output, skyloop_other, alloced_V4_array, etddim_array, k_array, Lsky, pixel_array, alloced_gpu, stream);
		for(int i=0; i<alloced_gpu; i++)				// wait for all commands in the stream to complete
			CUDA_CHECK(hipStreamSynchronize(stream[i]));
		finish[2] = clock();
		d[2] += (double)(finish[2] - start[2])/CLOCKS_PER_SEC;
		alloced_gpu = 0;
	}
	finish[0] = clock();
	
//	printf("diff time = %f\n", (double)(diff)/CLOCKS_PER_SEC);
	
	printf("time = %f\n", (double)((finish[0]-start[0])-d[1])/CLOCKS_PER_SEC);
	printf("d[2] = %f\n", d[2]);
	cleanup_cpu_mem(pre_gpu_data, post_gpu_data, stream);
	cleanup_gpu_mem(skyloop_output, skyloop_other, stream);
	for(int i=0; i<StreamNum; i++)	
		hipStreamDestroy(stream[i]);
	cout<<"Finish!"<<endl;

	return 0;
}

void QuickSort(size_t *V_array, int *k_array, int p, int r)
{
	int q;
	if(p<r)
	{
		q = Partition(V_array, k_array, p, r);
		QuickSort(V_array, k_array, p, q-1);
		QuickSort(V_array, k_array, q+1, r);
	}
}
int Partition(size_t *V_array, int *k_array, int p, int r)
{
	int x, i, j;
	int temp;
	x = V_array[k_array[r]];
	i = p-1;
	for(j = p; j<r; j++)
	{
		if(V_array[k_array[j]]<=x)
		{
			i = i + 1;
			temp = k_array[i];
			k_array[i] = k_array[j];
			k_array[j] = temp;
		}
	}
	temp = k_array[i+1];
	k_array[i+1] = k_array[r];
	k_array[r] = temp;
	i++;
	return i;
}
void CUDART_CB MyCallback(hipStream_t stream, hipError_t status, void* post_gpu_data)
{
	//FILE *fpt = fopen("./output/test_skyloopOutput1", "a");
	float aa;
	int l, lb;
	int le = 3071;
	lb = 0;
	int k;
	size_t V4;
	int count=0;
	size_t ptr = 0;
	cout<<"Callback"<<endl;
	k = ((post_data*)post_gpu_data)->other_data.k[count] - 1;
	while(k!=-1)
	{
		cout<<"callback k = "<<k<<endl;
		V4 = ((post_data*)post_gpu_data)->other_data.V4[count];
		ptr = ptr + 3072*V4;
		for(l=lb; l<=le; l++)
		{
		       	aa = ((post_data*)post_gpu_data)->output.output[ptr + l];
	//		fprintf(fpt, "k = %d l = %d aa = %f\n", k, l, aa);
		}
		ptr = ptr + 3072;
		count++;
		if(count<MaxPixel)
			k = ((post_data*)post_gpu_data)->other_data.k[count] - 1;
		else 
			break;
	}
	//fclose(fpt);
}


__host__ void push_work_into_gpu(struct pre_data *input_data, struct post_data *post_gpu_data, struct skyloop_output *skyloop_output, struct other *skyloop_other, size_t *alloced_V4_array, size_t *etddim_array, size_t k_array[][MaxPixel], int Lsky, int *pixel_array, int work_size, hipStream_t *stream)
{
	for(int i=0; i<work_size; i++)// transfer the data from CPU to GPU
	{
		hipMemcpyAsync(skyloop_other[i].eTD, input_data[i].other_data.eTD, etddim_array[i] * sizeof(float), hipMemcpyHostToDevice, stream[i] );
	}

	for(int i=0; i<work_size; i++)// call for gpu caculation
		kernel_skyloop<<<num_blocks, num_threads, shared_memory_usage, stream[i]>>>(skyloop_other[i].eTD, skyloop_other[0].ml_mm, skyloop_output[i].output, Lsky); 

	for(int i=0; i<work_size; i++)// transfer the data back from GPU to CPU
	{
        	hipMemcpyAsync(post_gpu_data[i].output.output, skyloop_output[i].output, Lsky * alloced_V4_array[i] * sizeof(float) + Lsky * pixel_array[i] * sizeof(float), hipMemcpyDeviceToHost, stream[i] );
	}
	for(int i=0; i<work_size; i++)
		hipStreamAddCallback(stream[i], MyCallback, (void*)&post_gpu_data[i], 0);
}

__global__ void kernel_skyloop(float *eTD, short *ml_mm, float *gpu_output, int Lsky) 
{
	const int grid_size = blockDim.x * gridDim.x;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	float *pe[NIFO];
	float *k_array;
	short *ml[NIFO];
	short *mm;
	size_t V, V4, tsize;
	int le = Lsky - 1;
	int l;
	int k;
	int msk;
	int count = 0;
	size_t etd_ptr = MaxPixel;
	size_t output_ptr = 0;
	k_array = eTD;
	
	k = k_array[count] - 1;
	ml[0] = ml_mm;
	ml[1] = ml_mm + Lsky;
	ml[2] = ml_mm + 2*Lsky;
	ml[3] = ml_mm + 3*Lsky;
	mm = ml_mm + 4*Lsky;
	while(k!=-1&&count<MaxPixel)
	{	
		V = constV[k];
		tsize = consttsize[k];
		msk = V%4;
		msk = (msk>0);
		V4 = V + msk*(4-V%4);
		
		pe[0] = eTD + etd_ptr;
		pe[1] = eTD + V4*tsize + etd_ptr;
		pe[2] = eTD + 2*V4*tsize + etd_ptr;
		pe[3] = eTD + 3*V4*tsize + etd_ptr;
	
		for(l = tid; l<=le; l+=grid_size)		// loop over sky locations
		{
			if(!mm[l]) continue;		// skip delay configurations
		
		// _sse_point_ps 
			pe[0] = pe[0] + (tsize/2)*V4;
			pe[1] = pe[1] + (tsize/2)*V4;
			pe[2] = pe[2] + (tsize/2)*V4;
			pe[3] = pe[3] + (tsize/2)*V4;
			pe[0] = pe[0] + ml[0][l] * (int)V4;
			pe[1] = pe[1] + ml[1][l] * (int)V4;
			pe[2] = pe[2] + ml[2][l] * (int)V4;
			pe[3] = pe[3] + ml[3][l] * (int)V4;
			
			/*if(k == 12 && l == 707)
			{
				gpu_output[l*V4] = pe[0][0];
				gpu_output[l*V4+1] = pe[1][0];
				gpu_output[l*V4+2] = pe[2][0];
				gpu_output[l*V4+3] = pe[3][0];
			}*/
		// inner skyloop
			kernel_skyloop_calculate(pe[0], pe[1], pe[2], pe[3], V, V4, V4*Lsky, gpu_output, l, output_ptr);
		}
			
		etd_ptr = etd_ptr + NIFO*V4*tsize;
		output_ptr = output_ptr + V4*Lsky + Lsky;
		count++;
		k = k_array[count] - 1;
	}
}

__inline__ __device__ void kernel_skyloop_calculate(float *PE_0, float *PE_1, float *PE_2, float *PE_3, size_t V, size_t V4, size_t rEDim, float *gpu_output, int l, size_t output_ptr) 
{
	int msk;						// mask
	size_t v = 0;					// indicate the pixel
	size_t ptr;						// indicate the location 
	float pe[NIFO];
	float _Eo[4], _Es[4], _En[4];
	float En, Es, Eo, aa;
	int Mm;
	float rE;						// energy array rNRG.data 
	float pE;						// energy array pNRG.data
	int count;
	
	//Eo = 0;							// total network energy
	//En = 0;							// network energy above the threshold
	//Es = 0;							// subnet energy above the threshold
	Mm = 0;							// # of pixels above the threshold
	
	for(count=0; count<4; count++)
	{
		_Eo[count] = 0;
		_Es[count] = 0;
		_En[count] = 0;
	}
	count = 0;
	ptr = l*V4 + output_ptr;
	while( v<V )					// loop over selected pixels	
	{
		// *_rE = _sse_sum_ps(_pe);
		pe[0] = PE_0[v];
		pe[1] = PE_1[v];
		pe[2] = PE_2[v];
		pe[3] = PE_3[v];
		rE = pe[0] + pe[1] + pe[2] + pe[3];								// get pixel energy
		//assign the value to the local memory
		gpu_output[ptr+v] = rE;
		msk = ( rE>=constEn );										// E>En  0/1 mask
		Mm += msk;												// count pixels above threshold
		///*new
		pE = rE * msk;											// zero sub-threshold pixels
		_Eo[count] += pE;												// network energy
		pE = kernel_minSNE_ps(pE, pe);						// subnetwork energy
		_Es[count] += pE;												// subnetwork energy
		msk = ( pE>=constEs );										// subnet energy > Es 0/1 mask
		rE *= msk;												
		_En[count] +=rE;											// network energy
		//En += rE;												// network energy
		// assign the value to the local memory
		v++;
		count++;
		count = count%4;
	}

	En = _En[0] + _En[1] + _En[2] + _En[3];												// Write back to output
	Eo = _Eo[0] + _Eo[1] + _Eo[2] + _Eo[3] + 0.01;												
	Es = _Es[0] + _Es[1] + _Es[2] + _Es[3];
	Mm = Mm *2 +0.01;
	aa = Es*En/(Eo-Es);
	
	msk = ((aa-Mm)/(aa+Mm)<0.33);								// if need continue 1/0
	aa = aa*(1-msk)  - 1*msk;
	gpu_output[rEDim + l + output_ptr] = aa;
}
__inline__ __device__ float kernel_minSNE_ps(float pE, float *pe)
{
	float a, b, c, d;
	int ab, ac, ad, bc, bd, cd;
	float temp;
	int flag;
	
	a = pe[0];
	b = pe[1];
	c = pe[2];
	d = pe[3];
	ab = ( a>=b );											// if a>=b, ab 1/0
	ac = ( a>=c );											// if a>=c, ac 1/0
	ad = ( a>=d );											// if a>=d, ad 1/0
	bc = ( b>=c );											// if b>=c, bc 1/0
	bd = ( b>=d );											// if b>=d, bd 1/0
	cd = ( c>=d );											// if c>=d, cd 1/0
	 
	temp = a+b+c+d - ab*ac*ad*a - (1-ab)*bc*bd*b - (1-ac)*(1-bc)*cd*c - (1-ad)*(1-bd)*(1-cd)*d;
	flag = ( temp>=pE );										// if temp>=pE, flag 1/0
	temp = temp + pE - flag*temp - (1-flag)*pE;
	return temp;
} 

void allocate_cpu_mem(struct pre_data *pre_gpu_data, struct post_data *post_gpu_data, int eTDDim, int V4max, int Lsky)// allocate locked memory on CPU 
{
	for(int i = 0; i<BufferNum; i++)
	{	
		CUDA_CHECK(hipHostAlloc(&(pre_gpu_data[i].other_data.eTD), NIFO * eTDDim * sizeof(float) + MaxPixel * sizeof(float), hipHostMallocMapped ) );
	}
	CUDA_CHECK(hipHostAlloc(&(pre_gpu_data[0].other_data.ml_mm), (1+NIFO) * Lsky * sizeof(short), hipHostMallocMapped ) );
	
	for(int i = 0; i<StreamNum; i++)
	{
		CUDA_CHECK(hipHostAlloc(&(post_gpu_data[i].output.output), Lsky * V4max * sizeof(float) + Lsky * sizeof(float), hipHostMallocMapped ) );
	}
	
		return;
}

void cleanup_cpu_mem(struct pre_data *pre_gpu_data, struct post_data *post_gpu_data, hipStream_t *stream)
{
 	for(int i = 0; i<BufferNum; i++)
	{
		CUDA_CHECK(hipHostFree(pre_gpu_data[i].other_data.eTD));
	}		
	CUDA_CHECK(hipHostFree(pre_gpu_data[0].other_data.ml_mm));
	for(int i=0; i<StreamNum; i++)
	{
		CUDA_CHECK(hipHostFree(post_gpu_data[i].output.output));
	}
	return;
}

void allocate_gpu_mem(struct skyloop_output *skyloop_output, struct other *skyloop_other, int eTDDim, int V4max, int Lsky)// allocate the memory on GPU
{
	for(int i = 0; i<StreamNum; i++)
	{
		CUDA_CHECK(hipMalloc(&(skyloop_other[i].eTD), NIFO * eTDDim * sizeof(float) ) );
		CUDA_CHECK(hipMalloc(&(skyloop_output[i].output), Lsky * V4max * sizeof(float) + Lsky * sizeof(float) ) );
	}
	CUDA_CHECK(hipMalloc(&(skyloop_other[0].ml_mm), (1+NIFO) * Lsky * sizeof(short) ) );	
}

void cleanup_gpu_mem(struct skyloop_output *skyloop_output, struct other *skyloop_other, hipStream_t *stream)// cleanup the memory on GPU
{
	for(int i = 0; i<StreamNum; i++)
	{
		CUDA_CHECK(hipFree(skyloop_other[i].eTD) );
		CUDA_CHECK(hipFree(skyloop_output[i].output) );
		//cout<<"cleanup gpu"<<endl;
	}
	CUDA_CHECK(hipFree(skyloop_other[0].ml_mm) );
	return;
}
