#include "hip/hip_runtime.h"
#include "/home/hpc/cWB/TEST/S6A_BKG_LF_L1H1V1_2G_SUPERCLUSTER_run1a_bench2/macro/gpu_struct.h"
#include "main.cuh"
#include <xmmintrin.h>
#include "wavearray.hh"
#include "gpu_network.hh"
//#include "/home/hpc/cWB/TEST/S6A_BKG_LF_L1H1V1_2G_SUPERCLUSTER_run1a_bench2/macro/gpu_function.h"
//#include "/home/hpc/cWB/trunk/wat/network.hh"
//#include "function.h"
/*#include "cwb.hh"
#include "cwb2G.hh"
#include "config.hh"
#include "network.hh"
#include "TString.h"
#include "TObjArray.h"
#include "TObjString.h"
#include "TRandom.h"
#include "TComplex.h"*/

#define num_blocks 16											// 16 blocks
#define num_threads 256											// 256 threads per block
#define shared_memory_usage 0										// no share memory

#define StreamNum 4 
#define BufferNum 4  
#define CONSTANT_SIZE 1500
#define MaxPixel 10


network *gpu_net;
TH2F *gpu_hist;
netcluster *pwc;
double *FP[NIFO];
double *FX[NIFO];
float *pa[StreamNum][NIFO][MaxPixel];
float *pA[StreamNum][NIFO][MaxPixel];
double gpu_d[10];
size_t gpu_nIFO;
size_t streamCount[StreamNum]; // the result of each stream

// GPU constant memory
__constant__ float constEn, constEs;	// two threshold
#define CUDA_CHECK(value) {                                             \
    hipError_t _m_cudaStat = value;                                    \
    if (_m_cudaStat != hipSuccess) {                                   \
        fprintf(stderr, "Error %s at line %d in file %s\n",             \
                hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);   \
        exit(1); }}

extern long Callback(void *post_gpu_data, network *gpu_net,  TH2F *gpu_hist, netcluster *pwc, double **FP, double **FX, float **pa, float **pA, size_t *streamCount, double *d);
long gpu_subNetCut(network *net, int lag, float snc, TH2F *hist, double *d)
{
	//define variables
	size_t nIFO = net->ifoList.size();
	
	float En = 2*net->acor*net->acor*nIFO;	// network energy threshold in the sky loop
	float Es = 2*net->e2or;			// subnet energy threshold in the sky loop
	float TH = fabs(snc);			// sub network threshold
	
	int l;
	float aa, AA;	
	size_t i, j, k, V, V4, id, K;
	int Lsky = int(net->index.size());
	short *mm = net->skyMask.data;
	
	short *ml[NIFO];

	clock_t start[10], finish[10];
	
	for(i=0; i<10; i++)
		gpu_d[i] = d[i];

	for( i=0; i<NIFO; i++)
	{
		if(i<nIFO)
		{
			ml[i] = net->getifo(i)->index.data;
			FP[i] = net->getifo(i)->fp.data;
			FX[i] = net->getifo(i)->fx.data;
		}
		else
		{
			ml[i] = net->getifo(0)->index.data;
			FP[i] = net->getifo(0)->fp.data;
			FX[i] = net->getifo(0)->fx.data;
		}
	}
	
	// allocate buffers
	std::vector<int> pI;			// buffer for pixel IDs	
	wavearray<double> cid;			// buffers for cluster ID
	netpixel *pix;	
	std::vector<int> *vint;
	pwc = &net->wc_List[lag];
	size_t count = 0;
	size_t tsize = 0;
	size_t V4max = 0;			// store the maximum of V4
	size_t Tmax = 0;			// store the maximum of tsize
	size_t *V_array, *tsize_array, *V4_array;
	int *k_sortArray;
	int kcount = 0;				// store the k that is not rejected/processed
	size_t k_array[StreamNum][MaxPixel];
	int CombineSize = V4max / 2;
	int etd_ptr, vtd_ptr, vTD_ptr;				// indicate the eTD's, vtd's and vTD's location
	size_t etddim_array[StreamNum];
	size_t alloced_V4_array[StreamNum];
	int pixel_array[StreamNum];
	int pixelCount;				// indicate the pixel number of each stream
	size_t alloced_V4;			// indicate the overall V4 of each stream
	int etddim;				// indicate the etd dim of each cluster;

//++++++++++++++++++++++++++++++++
// find out the maximum of V and tsize 
//++++++++++++++++++++++++++++++++

   	cid = pwc->get((char*)"ID",  0,'S',0);                  // get cluster ID

	K = cid.size();
	
	V_array = (size_t*)malloc(sizeof(size_t) * K);
	V4_array = (size_t*)malloc(sizeof(size_t) * K);
    	tsize_array = (size_t*)malloc(sizeof(size_t) * K);
    	k_sortArray = (int*)malloc(sizeof(int) * K);
	
	//cout<<"1"<<endl;
	for(k=0; k<K; k++)				// loop over clusters
	{
		V_array[k] = 0;
		tsize_array[k] = 0;
		id = size_t(cid.data[k]+0.1);
		if(pwc->sCuts[id-1] != -2) continue;	// skip rejected/processed culster
		vint = &(pwc->cList[id-1]);		// pixel list
		V = vint->size();			// pixel list size
		if(!V) continue;
		
		pI = net->wdmMRA.getXTalk(pwc, id);
	
		V = pI.size();				// number of loaded pixels
		if(!V) continue;
		
		pix = pwc->getPixel(id, pI[0]);
		tsize = pix->tdAmp[0].size();
		if(!tsize || tsize&1)			// tsize%1 = 1/0 = power/amplitude 
		{					 
			cout<<"network::subNetCut() error: wrong pixel TD data\n";
			exit(1);
		}
	
		tsize /= 2;
	    	V4 = V + (V%4 ? 4 - V%4 : 0);     
		V_array[k] = V;
		V4_array[k] = V4;
		tsize_array[k] = tsize;
		k_sortArray[kcount] = k;
		kcount++;
		
		if( tsize > Tmax )
			Tmax = tsize;
		if( V4 > V4max )
			V4max = V4;
	}
	
//++++++++++++++++++++++++++++++++
// declare the variables used for gpu calculation 
//++++++++++++++++++++++++++++++++
	struct pre_data pre_gpu_data[BufferNum];	// store the data before gpu cal
	struct post_data post_gpu_data[StreamNum];	// store the data transfer from gpu
	struct skyloop_output skyloop_output[StreamNum];// store the skyloop_output data
	struct other skyloop_other[StreamNum];		// store the data which is not output
	
	int eTDDim = 0;					// the size of each eTD
	int alloced_gpu = 0;				// the number of gpu which has been allocated data
	
	start[0] = clock();
	
	eTDDim = Tmax * V4max;
	for(int i=0; i<StreamNum; i++)
		streamCount[i] = 0;
	// allocate the memory on cpu and gpu
	allocate_cpu_mem(pre_gpu_data, post_gpu_data, eTDDim, V4max, Lsky);
	allocate_gpu_mem(skyloop_output, skyloop_other, eTDDim, V4max, Lsky);
	
	gpu_net = net;
	gpu_hist = hist;
	gpu_nIFO = nIFO;
	hipStream_t stream[StreamNum];			// define the stream
	for(int i=0; i<StreamNum; i++)			
		CUDA_CHECK(hipStreamCreate(&stream[i]));	// create the new stream
	hipMemcpyToSymbol(HIP_SYMBOL(constEn), &En, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(constEs), &Es, sizeof(float));

	for(int i=0; i<BufferNum; i++)		// initialize the data
	{
		for(int l=0; l<Lsky; l++)
		{
			for(int j=0; j<NIFO; j++)
			{
				int mlptr;
				mlptr = j*Lsky;
				post_gpu_data[i].other_data.ml_mm[mlptr + l] = ml[j][l];
			}
			post_gpu_data[i].other_data.ml_mm[NIFO*Lsky + l] = mm[l];
		}
		post_gpu_data[i].other_data.T_En = En;
		post_gpu_data[i].other_data.T_Es = Es;
		post_gpu_data[i].other_data.TH = TH;
		post_gpu_data[i].other_data.le = Lsky - 1;
		post_gpu_data[i].other_data.lag = lag;
		post_gpu_data[i].other_data.nIFO = nIFO;
	}
	for(int l=0; l<Lsky; l++)
	{
		for(int j=0; j<NIFO; j++)
		{
			int mlptr;
			mlptr = j*Lsky;
			pre_gpu_data[0].other_data.ml_mm[mlptr + l] = ml[j][l];	
		}
		pre_gpu_data[0].other_data.ml_mm[NIFO*Lsky+ l] = mm[l];
	}
	hipMemcpyAsync(skyloop_other[0].ml_mm, pre_gpu_data[0].other_data.ml_mm, (1 + NIFO) * Lsky * sizeof(short), hipMemcpyHostToDevice, stream[0] );
	finish[0] = clock();
	d[3] += (double)(finish[0] - start[0])/CLOCKS_PER_SEC;
//++++++++++++++++++++++++++++++++
// loop over cluster
//++++++++++++++++++++++++++++++++
	std::vector<wavearray<float> > vtd;     // vectors of TD energies  
     	std::vector<wavearray<float> > vTD;     // vectors of TD energies  
	wavearray<float> tmp(Tmax*V4max); tmp=0;  // aligned array for TD amplitude	

	QuickSort(V_array, k_sortArray, 0, kcount-1);
   	cid = pwc->get((char*)"ID",  0,'S',0);                 // get cluster ID
   	K = cid.size();                                                         
	
	start[1] = clock();
	//cout<<"2"<<endl;
	alloced_V4 = 0;						// initialize
	etd_ptr = MaxPixel;
	vtd_ptr = 0;
	vTD_ptr = 0;
	pixelCount = 0;
	for(i=0; i<NIFO; i++)
	{
		vtd.push_back(tmp);
		vTD.push_back(tmp);
	}
	for(int z=0; z<kcount; z++)				// loop over clusters
	{
		start[2] = clock();
		k = k_sortArray[z];
		V4 = V4_array[k];
		tsize = tsize_array[k];
		etddim = V4 * tsize;
		alloced_V4 += V4;
		if(alloced_V4<=V4max && (alloced_V4 + pixelCount + 1)<=(V4max+1))
		{
			id = size_t(cid.data[k]+0.1);
			pI = net->wdmMRA.getXTalk(pwc, id);
	
			for(i=0; i<NIFO; i++)
			{
				pa[alloced_gpu][i][pixelCount] = vtd[i].data + (tsize/2)*V4 + vtd_ptr;
				pA[alloced_gpu][i][pixelCount] = vTD[i].data + (tsize/2)*V4 + vTD_ptr;
			}

			for( j=0; j<V; j++)
			{  
				pix = pwc->getPixel(id,pI[j]);          // get pixel pointer   
				for(i=0; i<nIFO; i++) 
				{
            				for( l=0; l<tsize; l++) 
					{                                
	              		   		aa = pix->tdAmp[i].data[l];             // copy TD 00 data 
				        	AA = pix->tdAmp[i].data[l+tsize];       // copy TD 90 data 
						vtd[i].data[l*V4+j+vtd_ptr] = aa;
						vTD[i].data[l*V4+j+vTD_ptr] = AA;
						// assign the data 
					   	if(alloced_gpu<BufferNum)
				  		{
				   			pre_gpu_data[alloced_gpu].other_data.eTD[i*etddim + l*V4+j + etd_ptr] = aa*aa+AA*AA;
							if(i == nIFO - 1 && NIFO > nIFO)
								for(int I = nIFO; I<NIFO; I++)
					   				pre_gpu_data[alloced_gpu].other_data.eTD[I*etddim + l*V4+j + etd_ptr] = 0;
					   	}	
        	    			}
				}
			}
			i = alloced_gpu;
			etd_ptr += NIFO*etddim;
			k_array[i][pixelCount] = k+1;
			pre_gpu_data[i].other_data.eTD[pixelCount] = k+1;
			post_gpu_data[i].other_data.k[pixelCount] = k+1;
			post_gpu_data[i].other_data.V[pixelCount] = V_array[k];
			post_gpu_data[i].other_data.V4[pixelCount] = V4;
			post_gpu_data[i].other_data.tsize[pixelCount] = tsize;
			post_gpu_data[i].other_data.id[pixelCount] = id;
			pixelCount++;
		}
		finish[2] = clock();
		d[5] += (double)(finish[2] - start[2])/CLOCKS_PER_SEC;
		if( pixelCount<MaxPixel && alloced_V4<CombineSize )	continue;
		else if(alloced_V4>V4max || (alloced_V4 + pixeCount)>(V4max+1))
		{
			alloced_V4 -= V4_array[k];
			count--;
		}
		cout<<"list"<<i<<" overall V4 = "<<alloced_V4<<endl;
		for(int z=0; z<pixelCount; z++)
			cout<<"k = "<<pre_gpu_data[i].other_data.eTD[z]-1<<endl;
		etddim_array[i] = etd_ptr;
		alloced_V4_array[i] = alloced_V4;	
		pixel_array[i] = pixelCount;
		alloced_gpu++;
//++++++++++++++++++++++++++++++++
// assign the data 
//++++++++++++++++++++++++++++++++
		if(alloced_gpu == StreamNum)	// if all streams' data have been assigned
		{
			start[3] = clock();
			post_gpu_data[i].other_data.stream = i;
			push_work_into_gpu(pre_gpu_data, post_gpu_data, skyloop_output, skyloop_other, alloced_V4_array, etddim_array, k_array, Lsky, pixel_array, StreamNum, stream);
			for(int i=0; i<StreamNum; i++)				// wait for all commands in the stream to complete
				CUDA_CHECK(hipStreamSynchronize(stream[i]));
			alloced_gpu = 0;
			finish[3] =  clock();
			d[6] += (double)(finish[3] - start[3])/CLOCKS_PER_SEC;
			for(j=0; j<StreamNum; j++)
				for(i=0; i<pixel_array[j]; i++)
				{
					k_array[j][i] = 0;
					post_gpu_data[j].other_data.id[i] = 0;
					post_gpu_data[j].other_data.k[i] = 0;
					post_gpu_data[j].other_data.V4[i] = 0;
					post_gpu_data[j].other_data.tsize[i] = 0;
				}
		}
		//clear
		etd_ptr = MaxPixel;
		vtd_ptr = 0;
		vTD_ptr = 0;
		pixelCount = 0;
		alloced_V4 = 0;
	}							// end of loop
	if(alloced_gpu != 0)		// if there are some clusters waiting for GPU calculation
	{	
		push_work_into_gpu(pre_gpu_data, post_gpu_data, skyloop_output, skyloop_other, V_array, tsize_array, k_array, Lsky, alloced_gpu, stream);
		for(int i=0; i<alloced_gpu; i++)				// wait for all commands in the stream to complete
			CUDA_CHECK(hipStreamSynchronize(stream[i]));
		alloced_gpu = 0;
	}
	finish[1] = clock();
	d[4] += (double)(finish[1] - start[1])/CLOCKS_PER_SEC;
	cleanup_cpu_mem(pre_gpu_data, post_gpu_data, stream);
	cleanup_gpu_mem(skyloop_output, skyloop_other, stream);
	for(int i=0; i<StreamNum; i++)	
		hipStreamDestroy(stream[i]);
	for(int i=0; i<StreamNum; i++)
		count += streamCount[i];
	cout<<"count = "<<count<<endl;
	for(i=0; i<3; i++)
		d[i] = gpu_d[i];
	return count;
}

__host__ void push_work_into_gpu(struct pre_data *input_data, struct post_data *post_gpu_data, struct skyloop_output *skyloop_output, struct other *skyloop_other, size_t *alloced_V4_array, size_t *etddim_array, size_t k_array[][MaxPixel], int Lsky, int *pixel_array, int work_size, hipStream_t *stream)
{
	for(int i=0; i<work_size; i++)// transfer the data from CPU to GPU
		hipMemcpyAsync(skyloop_other[i].eTD, input_data[i].other_data.eTD, etddim_array[i] * sizeof(float), hipMemcpyHostToDevice, stream[i] );

	for(int i=0; i<work_size; i++)// call for gpu caculation
		kernel_skyloop<<<num_blocks, num_threads, shared_memory_usage, stream[i]>>>(skyloop_other[i].eTD, skyloop_other[0].ml_mm, skyloop_output[i].output, Lsky);
	for(int i=0; i<work_size; i++)// transfer the data back from GPU to CPU
        	hipMemcpyAsync(post_gpu_data[i].output.output, skyloop_output[i].output, Lsky * alloced_V4_array[i] * sizeof(float) + Lsky * pixel_array[i] * sizeof(float), hipMemcpyDeviceToHost, stream[i] );
	for(int i=0; i<work_size; i++)
		hipStreamAddCallback(stream[i], MyCallback, (void*)&post_gpu_data[i], 0);
}

__global__ void kernel_skyloop(float *eTD, short *ml_mm, float *gpu_output, int Lsky) 
{
	const int grid_size = blockDim.x * gridDim.x;
	int l = blockIdx.x * blockDim.x + threadIdx.x;
	float *pe[NIFO];
	short *ml[NIFO];
	short *mm;
	int msk;	
	size_t V4;
	int le = Lsky - 1;

	msk = V%4;
	msk = (msk>0);
	V4 = V + msk*(4-V%4);

	pe[0] = eTD;
	pe[1] = eTD + V4*tsize;
	pe[2] = eTD + 2*V4*tsize;
	pe[3] = eTD + 3*V4*tsize;
	ml[0] = ml_mm;
	ml[1] = ml_mm + Lsky;
	ml[2] = ml_mm + 2*Lsky;
	ml[3] = ml_mm + 3*Lsky;
	mm = ml_mm + 4*Lsky;

	for(; l<=le; l+=grid_size)		// loop over sky locations
	{
		if(!mm[l])	continue;
		// _sse_point_ps 
		pe[0] = pe[0] + (tsize/2)*V4;
		pe[1] = pe[1] + (tsize/2)*V4;
		pe[2] = pe[2] + (tsize/2)*V4;
		pe[3] = pe[3] + (tsize/2)*V4;

		pe[0] = pe[0] + ml[0][l] * (int)V4;
		pe[1] = pe[1] + ml[1][l] * (int)V4;
		pe[2] = pe[2] + ml[2][l] * (int)V4;
		pe[3] = pe[3] + ml[3][l] * (int)V4;
		
		// inner skyloop
		kernel_skyloop_calculate(pe[0], pe[1], pe[2], pe[3], V, V4, V4*Lsky, gpu_output, l);
	}
		
}

__inline__ __device__ void kernel_skyloop_calculate(float *PE_0, float *PE_1, float *PE_2, float *PE_3, size_t V, size_t V4, size_t rEDim, float *gpu_output,  int l) 
{
	float msk;						// mask
	int v;					// indicate the pixel
	int ptr;						// indicate the location 
	float pe[NIFO];
	float _Eo[4], _Es[4], _En[4];
	float En, Es, Eo, aa;
	int Mm;
	float rE;						// energy array rNRG.data 
	float pE;						// energy array pNRG.data
	int count;
	
	Mm = 0;							// # of pixels above the threshold

	for(count=0; count<4; count++)
	{
		_Eo[count] = 0;
		_Es[count] = 0;
		_En[count] = 0;
	}
	
	ptr = l*(int)V4;
	count = 0;
	for(v=0; v<V; v++)					// loop over selected pixels	
	{
		// *_rE = _sse_sum_ps(_pe);
		pe[0] = PE_0[v];
		pe[1] = PE_1[v];
		pe[2] = PE_2[v];
		pe[3] = PE_3[v];
		rE = pe[0] + pe[1] + pe[2] + pe[3];								// get pixel energy
		//assign the value to the local memory
		gpu_output[ptr + v] = rE;
      	// E>En  0/1 mask
		msk = ( rE>=constEn );										// E>En  0/1 mask
		Mm += msk;												// count pixels above threshold
		pE = rE * msk;											// zero sub-threshold pixels
		_Eo[count] += pE;
		//Eo += pE;												// network energy
		pE = kernel_minSNE_ps(pE, pe);						// subnetwork energy
		_Es[count] += pE;
		//Es += pE;												// subnetwork energy
		msk = ( pE>=constEs );										// subnet energy > Es 0/1 mask
		rE *= msk;											   
		_En[count] += rE;
		// assign the value to the local memory
		count++;
		count = count%4;
	}

	En = _En[0] + _En[1] + _En[2] + _En[3];			// Write back to output
	Eo = _Eo[0] + _Eo[1] + _Eo[2] + _Eo[3] + 0.01;
	Es = _Es[0] + _Es[1] + _Es[2] + _Es[3];
	Mm = Mm * 2 + 0.01;
	aa = Es*En/(Eo-Es);
	
	msk = ((aa-Mm)/(aa+Mm)<0.33);
	aa = aa*(1-msk) + (-1)*msk;
	gpu_output[rEDim + l] = aa;
	
}
__inline__ __device__ float kernel_minSNE_ps(float pE, float *pe)
{
	float a, b, c, d;
	int ab, ac, ad, bc, bd, cd;
	float temp;
	int flag;
	
	a = pe[0];
	b = pe[1];
	c = pe[2];
	d = pe[3];
	ab = ( a>=b );											// if a>=b, ab 1/0
	ac = ( a>=c );											// if a>=c, ac 1/0
	ad = ( a>=d );											// if a>=d, ad 1/0
	bc = ( b>=c );											// if b>=c, bc 1/0
	bd = ( b>=d );											// if b>=d, bd 1/0
	cd = ( c>=d );											// if c>=d, cd 1/0
	 
	temp = a+b+c+d - ab*ac*ad*a - (1-ab)*bc*bd*b - (1-ac)*(1-bc)*cd*c - (1-ad)*(1-bd)*(1-cd)*d;
	flag = ( temp>=pE );										// if temp>=pE, flag 1/0
	temp = (temp - flag*temp) + (pE - (1-flag)*pE);
	return temp;
}
 
void CUDART_CB MyCallback(hipStream_t stream, hipError_t status, void* post_gpu_data)
{
	int streamNum;
	streamNum = ((post_data*)post_gpu_data)->other_data.stream;
	//cout<<"Callback"<<endl;
	Callback(post_gpu_data, gpu_net, gpu_hist, pwc, FP, FX, pa[streamNum], pA[streamNum], streamCount, gpu_d);
}

void allocate_cpu_mem(struct pre_data *pre_gpu_data, struct post_data *post_gpu_data, int eTDDim, int V4max, int Lsky)// allocate locked memory on CPU 
{
	for(int i = 0; i<BufferNum; i++)
	{	

		CUDA_CHECK(hipHostAlloc(&(pre_gpu_data[i].other_data.eTD), NIFO * eTDDim * sizeof(float), hipHostMallocMapped ) );
	}
	CUDA_CHECK(hipHostAlloc(&(pre_gpu_data[0].other_data.ml_mm), (1 + NIFO) * Lsky * sizeof(short), hipHostMallocMapped ) );
	for(int i = 0; i<StreamNum; i++)
	{
		CUDA_CHECK(hipHostAlloc(&(post_gpu_data[i].output.output), Lsky * V4max * sizeof(float) + Lsky * sizeof(float), hipHostMallocMapped ) );
		post_gpu_data[i].other_data.ml_mm = (short*)malloc(sizeof(size_t) * (1 + NIFO) * Lsky);
	}
		return;
}

void cleanup_cpu_mem(struct pre_data *pre_gpu_data, struct post_data *post_gpu_data, hipStream_t *stream)
{
 	for(int i = 0; i<BufferNum; i++)
	{
		CUDA_CHECK(hipHostFree(pre_gpu_data[i].other_data.eTD));
	}		
	CUDA_CHECK(hipHostFree(pre_gpu_data[0].other_data.ml_mm));
	for(int i=0; i<StreamNum; i++)
	{
		CUDA_CHECK(hipHostFree(post_gpu_data[i].output.output));
		free(post_gpu_data[i].other_data.ml_mm);
	}
	return;
}

void allocate_gpu_mem(struct skyloop_output *skyloop_output, struct other *skyloop_other, int eTDDim, int V4max, int Lsky)// allocate the memory on GPU
{
	for(int i = 0; i<StreamNum; i++)
	{
		CUDA_CHECK(hipMalloc(&(skyloop_other[i].eTD), NIFO * eTDDim * sizeof(float) ) );
		CUDA_CHECK(hipMalloc(&(skyloop_output[i].output), Lsky * V4max * sizeof(float) + Lsky * sizeof(float) ) );
	}
	CUDA_CHECK(hipMalloc(&(skyloop_other[0].ml_mm), (1 + NIFO) * Lsky * sizeof(short) ) );	
}

void cleanup_gpu_mem(struct skyloop_output *skyloop_output, struct other *skyloop_other, hipStream_t *stream)// cleanup the memory on GPU
{
	for(int i = 0; i<StreamNum; i++)
	{
		CUDA_CHECK(hipFree(skyloop_other[i].eTD) );
		CUDA_CHECK(hipFree(skyloop_output[i].output) );
	}
	CUDA_CHECK(hipFree(skyloop_other[0].ml_mm) );
	return;
}

void QuickSort(size_t *V_array, int *k_array, int p, int r)
{
        int q;
        if(p<r)
        {
                q = Partition(V_array, k_array, p, r);
                QuickSort(V_array, k_array, p, q-1);
                QuickSort(V_array, k_array, q+1, r);
        }
}
int Partition(size_t *V_array, int *k_array, int p, int r)
{
        int x, i, j;
        int temp;
        x = V_array[k_array[r]];
        i = p-1;
        for(j = p; j<r; j++)
        {
                if(V_array[k_array[j]]<=x)
                {
                        i = i + 1;
                        temp = k_array[i];
                        k_array[i] = k_array[j];
                        k_array[j] = temp;
                }
        }
        temp = k_array[i+1];
        k_array[i+1] = k_array[r];
        k_array[r] = temp;
        i++;
        return i;
}
